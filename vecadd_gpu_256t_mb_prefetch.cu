#include "hip/hip_runtime.h"
//
// Created by Elliot Warren on 12/1/2023.
// Bulk of the code copied from https://developer.nvidia.com/blog/even-easier-introduction-cuda/
// edits made include addition of timer, calculations for time, MFLOPs, memory bandwidth
// also tweaked N value.
//

#include <cmath>
#include <iostream>
#include <iomanip>

// function to add the elements of two arrays
//global specifier to add code to gpu kernel
__global__
void add(int n, const float *x, float *y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}

int main()
{
    //from https://www.amd.com/en/products/cpu/amd-epyc-7763
    //double capacity = 204.8e9; // Peak memory bandwidth in bytes/sec.

    int deviceID = 0;
    int N = 1<<29; // 512M elements
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    std::cout << "number of blocks: " << numBlocks << std::endl;
    float *x, *y;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    hipMemPrefetchAsync((void *)x, N*sizeof(float), deviceID);
    hipMemPrefetchAsync((void *)y, N*sizeof(float), deviceID);

    // Run kernel on 512M elements on the GPU
    add<<<numBlocks, blockSize>>>(N, x, y);

    /*double duration = std::chrono::duration<double>(end - start).count();

    double mflops = (N/1e6) / duration;

    double bytes = N * sizeof(uint64_t);

    double memoryBandwidthUtilized = ((((bytes / 1e9) / duration) / capacity) * 100) * 1e9; // % of memory bandwidth utilized

    printf("Time elapsed: %f seconds\n", duration);
    printf("MFLOP/s: %f\n", mflops);
    printf("% Memory bandwidth utilized: %f\n", memoryBandwidthUtilized);*/

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = std::fmax(maxError, std::fabs(y[i]-3.0f));
    std::cout << "Max error: " << maxError << std::setprecision(5) << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}